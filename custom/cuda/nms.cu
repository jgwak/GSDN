#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
// Modified and redistributed by JunYoung Gwak
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>

#include <vector>
#include <iostream>

int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  float xmin = max(a[0], b[0]), xmax = min(a[3], b[3]);
  float ymin = max(a[1], b[1]), ymax = min(a[4], b[4]);
  float zmin = max(a[2], b[2]), zmax = min(a[5], b[5]);
  float xsize = max(xmax - xmin, 0.f), ysize = max(ymax - ymin, 0.f);
  float zsize = max(zmax - zmin, 0.f);
  float interS = xsize * ysize * zsize;
  float Sa = (a[3] - a[0]) * (a[4] - a[1]) * (a[5] - a[2]);
  float Sb = (b[3] - b[0]) * (b[4] - b[1]) * (b[5] - b[2]);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 7];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 7 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 0];
    block_boxes[threadIdx.x * 7 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 1];
    block_boxes[threadIdx.x * 7 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 2];
    block_boxes[threadIdx.x * 7 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 3];
    block_boxes[threadIdx.x * 7 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 4];
    block_boxes[threadIdx.x * 7 + 5] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 5];
    block_boxes[threadIdx.x * 7 + 6] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 6];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 7;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 7) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = THCCeilDiv(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

// boxes is a N x 7 tensor
at::Tensor nms_cuda(const at::Tensor boxes, float nms_overlap_thresh) {
  using scalar_t = float;
  AT_ASSERTM(boxes.type().is_cuda(), "boxes must be a CUDA tensor");
  auto scores = boxes.select(1, 6);
  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t);

  int boxes_num = boxes.size(0);

  const int col_blocks = THCCeilDiv(boxes_num, threadsPerBlock);

  scalar_t* boxes_dev = boxes_sorted.data<scalar_t>();

  THCState *state = at::globalContext().lazyInitCUDA(); // TODO replace with getTHCState

  unsigned long long* mask_dev = NULL;
  //THCudaCheck(THCudaMalloc(state, (void**) &mask_dev,
  //                      boxes_num * col_blocks * sizeof(unsigned long long)));

  mask_dev = (unsigned long long*) THCudaMalloc(state, boxes_num * col_blocks * sizeof(unsigned long long));

  dim3 blocks(THCCeilDiv(boxes_num, threadsPerBlock),
              THCCeilDiv(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  THCudaCheck(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);
  std::vector<unsigned long long> remi(boxes_num);
  memset(&remi[0], -1, sizeof(unsigned long long) * boxes_num);

  at::Tensor keep = at::empty({boxes_num}, boxes.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data<int64_t>();

  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[i] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        unsigned long long is_new_overlap = p[j] & ~remv[j];
        int start_thread;
        if (j == nblock) {
            start_thread = inblock + 1;
        } else {
            start_thread = 0;
        }
        for (int k = start_thread; k < threadsPerBlock; k++) {
            if(is_new_overlap & (1ULL << k)) {
                remi[j * threadsPerBlock + k] = i;
            }
        }
        remv[j] |= p[j];
      }
    } else {
        keep_out[i] = remi[i];
    }
  }

  THCudaFree(state, mask_dev);
  return order_t.index({keep.to(order_t.device(), keep.scalar_type())});
}
